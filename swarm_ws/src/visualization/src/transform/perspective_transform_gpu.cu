#include "hip/hip_runtime.h"
#include "perspective_transform_gpu.h"
#include "../inc/hip/hip_runtime_api.h"
#include "../inc/helper_functions.h"

#include <unistd.h>

__device__ double scale(double val, double o_min, double o_max, double n_min,
		double n_max)
{
	if (o_max == o_min) // special case
		return n_min;
	return ((val - o_min) / (o_max - o_min)) * (n_max - n_min) + n_min; // shifting numeric domains
}

__device__ vector2f_t warpPoint(quadrilateral_t trap, size_t width,
		size_t height, vector2f_t initial)
{
	// finding connected edges
	vector2f_t top = {scale((float)initial.x, 0.0f, (float)width, (float)trap.tl.x, (float)trap.tr.x),
			scale((float)initial.x, 0.0f,(float) width, (float)trap.tl.y, (float)trap.tr.y)};
	vector2f_t bottom = {scale((float)initial.x, 0.0f, (float)width, (float)trap.bl.x, (float)trap.br.x),
			scale((float)initial.x, 0.0f, (float)width, (float)trap.bl.y, (float)trap.br.y)};
	vector2f_t left = {scale((float)initial.y, 0.0f, (float)height, (float)trap.bl.x, (float)trap.tl.x),
			scale((float)initial.y, 0.0f, (float)height, (float)trap.bl.y, (float)trap.tl.y)};
	vector2f_t right = {scale((float)initial.y, 0.0f, (float)height, (float)trap.br.x, (float)trap.tr.x),
			scale((float)initial.y, 0.0f, (float)height, (float)trap.br.y, (float)trap.tr.y)};

	// linear intersection
	double m0 = (right.y - left.y) / (right.x - left.x);
	double m1 = (bottom.y - top.y) / (bottom.x - top.x);
	double unified_x =
			top.x != bottom.x && m0 != m1 && left.x != right.x ?
					(top.y - right.y + right.x * m0 - top.x * m1) / (m0 - m1) : top.x;
	double unified_y =
			left.y != right.y ? (m0 * (unified_x - right.x) + right.y) : left.y;

	return (vector2f_t){(float) unified_x, (float) unified_y};
}

__global__ void transform(sf::Uint8 *cols_in, sf::Uint8 *cols_out, size_t *width, size_t *height, quadrilateral_t *trap)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x; // finding iteration level
	int row = i / *width;
	int col = i % *width;
	vector2f_t square_pos = {(float) col, (float) row}; // creating vector of the current position in the rectangle
	vector2f_t trap_point = warpPoint(*trap, *width, *height, square_pos); // finding corresponding position
	if ((int) trap_point.x < *width && (int) trap_point.y < *height
			&& (int) trap_point.x >= 0 && (int) trap_point.y >= 0) // safety catch for drawing pixels
	{
		// copying pixel from one frame to another
		int col_ind = i * 4;
		int o_col_ind = ((int)trap_point.y * (*width) + (int)trap_point.x) * 4;
		for (size_t j = 0;j < 4;j++) // getting values for RGBA
		{
			cols_out[o_col_ind + j] = cols_in[col_ind + j];
		}
	}
}

void createDeviceVar(void **var, size_t size, void *h_var)
{
    std::cout << "\033[31;1m" << std::flush;
    checkCudaErrors(hipMalloc(var, size)); // allocating device memory
    checkCudaErrors(hipMemset(*var, 0, size)); // writing 0s to location
    std::cout << "\033[0m" << std::flush;

    hipMemcpyAsync(*var, h_var, size, hipMemcpyHostToDevice, 0); // copying from host
}

void perspectiveTransform(quadrilateral_t trap, sf::RenderTexture *rt, sf::Uint8 *tf_cols)
{
	// setting up variables that can be passed
	sf::Image img = rt->getTexture().copyToImage(); // getting image
	sf::Uint8 *col_ptr = (sf::Uint8 *)img.getPixelsPtr(); // getting pixel array
	size_t width = img.getSize().x; // getting dimensions
	size_t height = img.getSize().y;

	// copying to calculation
	// device variables
	sf::Uint8 *d_col_in;
	sf::Uint8 *d_col_out;
	size_t *d_width;
	size_t *d_height;
	quadrilateral_t *d_trap;

	// copying data over
	createDeviceVar((void **)&d_col_in, 4 * sizeof(sf::Uint8) * width * height, col_ptr);

	// out does not have an initial host variable and was merely set up
	checkCudaErrors(hipMalloc((void **)&d_col_out, 4 * sizeof(sf::Uint8) * width * height));
	checkCudaErrors(hipMemset(d_col_out, 0, 4 * sizeof(sf::Uint8) * width * height));

	createDeviceVar((void **)&d_width, sizeof(size_t), &width);
	createDeviceVar((void **)&d_height, sizeof(size_t), &height);
	createDeviceVar((void **)&d_trap, sizeof(quadrilateral_t), &trap);

	// setting up processing specifics
	dim3 threads(512, 1); // number of threads per block
	dim3 blocks(width * height / threads.x, 1); // number of blocks in data set to be calculated

	// setup start stop events
	hipEvent_t start, stop;
	checkCudaErrors(hipEventCreate(&start));
	checkCudaErrors(hipEventCreate(&stop));

	hipEventRecord(start, 0); // start event trigger
	transform<<<blocks, threads>>>(d_col_in, d_col_out, d_width, d_height, d_trap); // perform calculation
	hipMemcpyAsync(tf_cols, d_col_out, 4 * sizeof(sf::Uint8) * width * height, hipMemcpyDeviceToHost, 0); // copy useful data back

	hipEventRecord(stop, 0); // stop event trigger

	while (hipEventQuery(stop) == hipErrorNotReady) // wait for stop
	{
			usleep(100);
	}

	// cleaning up
	checkCudaErrors(hipEventDestroy(start));
	checkCudaErrors(hipEventDestroy(stop));
	checkCudaErrors(hipFree(d_col_in));
	checkCudaErrors(hipFree(d_col_out));
	checkCudaErrors(hipFree(d_width));
	checkCudaErrors(hipFree(d_height));
	checkCudaErrors(hipFree(d_trap));
}
