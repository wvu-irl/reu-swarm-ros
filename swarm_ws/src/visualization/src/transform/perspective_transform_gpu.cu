#include "hip/hip_runtime.h"
/*********************************************************************
* Software License Agreement (BSD License)
*
* Copyright (c) 2019, WVU Interactive Robotics Laboratory
*                       https://web.statler.wvu.edu/~irl/
* All rights reserved.
*
*  Redistribution and use in source and binary forms, with or without
*  modification, are permitted provided that the following conditions
*  are met:
*
*   * Redistributions of source code must retain the above copyright
*     notice, this list of conditions and the following disclaimer.
*   * Redistributions in binary form must reproduce the above
*     copyright notice, this list of conditions and the following
*     disclaimer in the documentation and/or other materials provided
*     with the distribution.
*   * Neither the name of the Willow Garage nor the names of its
*     contributors may be used to endorse or promote products derived
*     from this software without specific prior written permission.
*
*  THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS
*  "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT
*  LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS
*  FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE
*  COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT,
*  INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
*  BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
*  LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
*  CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT
*  LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN
*  ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
*  POSSIBILITY OF SUCH DAMAGE.
*********************************************************************/

#include <visualization/perspective_transform_gpu.h>
#include "../inc/hip/hip_runtime_api.h"
#include "../inc/helper_functions.h"

#include <unistd.h>

__device__ double scale(double val, double o_min, double o_max, double n_min,
		double n_max)
{
	if (o_max == o_min) // special case
		return n_min;
	return ((val - o_min) / (o_max - o_min)) * (n_max - n_min) + n_min; // shifting numeric domains
}

__device__ vector2f_t warpPoint(quadrilateral_t trap, size_t width,
		size_t height, vector2f_t initial)
{
	// finding connected edges
	vector2f_t top = {(float)scale(initial.x, 0.0, width, trap.tl.x, trap.tr.x),
			(float)scale(initial.x, 0.0, width, trap.tl.y, trap.tr.y)};
	vector2f_t bottom = {(float)scale(initial.x, 0.0, width, trap.bl.x, trap.br.x),
			(float)scale(initial.x, 0.0, width, trap.bl.y, trap.br.y)};
	vector2f_t left = {(float)scale(initial.y, 0.0, height, trap.bl.x, trap.tl.x),
			(float)scale(initial.y, 0.0, height, trap.bl.y, trap.tl.y)};
	vector2f_t right = {(float)scale(initial.y, 0.0, height, trap.br.x, trap.tr.x),
			(float)scale(initial.y, 0.0, height, trap.br.y, trap.tr.y)};

	// linear intersection
	double m0 = (right.y - left.y) / (right.x - left.x);
	double m1 = (bottom.y - top.y) / (bottom.x - top.x);
	double unified_x =
			top.x != bottom.x && m0 != m1 && left.x != right.x ?
					(top.y - right.y + right.x * m0 - top.x * m1) / (m0 - m1) : top.x;
	double unified_y =
			left.y != right.y ? (m0 * (unified_x - right.x) + right.y) : left.y;

	return (vector2f_t){(float) unified_x, (float) unified_y};
}

__global__ void transform(sf::Uint8 *cols_in, sf::Uint8 *cols_out, size_t *width, size_t *height, quadrilateral_t *trap)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x; // finding iteration level
	int row = *height - i / *width;
	int col = i % *width;
	vector2f_t square_pos = {(float) col, (float) row}; // creating vector of the current position in the rectangle
	vector2f_t trap_point = warpPoint(*trap, *width, *height, square_pos); // finding corresponding position
	if ((int) trap_point.x < *width && (int) trap_point.y < *height
			&& (int) trap_point.x >= 0 && (int) trap_point.y >= 0) // safety catch for drawing pixels
	{
		// copying pixel from one frame to another
		int col_ind = i * 4;
		int o_col_ind = ((int)trap_point.y * (*width) + (int)trap_point.x) * 4;

		for (size_t j = 0;j < 4;j++) // getting values for RGBA
		{
			cols_out[o_col_ind + j] = cols_in[col_ind + j];
		}
	}
}

void createDeviceVar(void **var, size_t size, void *h_var)
{
    std::cout << "\033[31;1m" << std::flush;
    checkCudaErrors(hipMalloc(var, size)); // allocating device memory
    checkCudaErrors(hipMemset(*var, 0, size)); // writing 0s to location
    std::cout << "\033[0m" << std::flush;

    hipMemcpyAsync(*var, h_var, size, hipMemcpyHostToDevice, 0); // copying from host
}

void perspectiveTransform(quadrilateral_t trap, sf::RenderTexture *rt, sf::Uint8 *tf_cols)
{
	// setting up variables that can be passed
	sf::Image img = rt->getTexture().copyToImage(); // getting image
	sf::Uint8 *col_ptr = (sf::Uint8 *)img.getPixelsPtr(); // getting pixel array
	size_t width = img.getSize().x; // getting dimensions
	size_t height = img.getSize().y;

	// copying to calculation
	// device variables
	sf::Uint8 *d_col_in;
	sf::Uint8 *d_col_out;
	size_t *d_width;
	size_t *d_height;
	quadrilateral_t *d_trap;

	// copying data over
	createDeviceVar((void **)&d_col_in, 4 * sizeof(sf::Uint8) * width * height, col_ptr);

	// out does not have an initial host variable and was merely set up
	checkCudaErrors(hipMalloc((void **)&d_col_out, 4 * sizeof(sf::Uint8) * width * height));
	checkCudaErrors(hipMemset(d_col_out, 0, 4 * sizeof(sf::Uint8) * width * height));

	createDeviceVar((void **)&d_width, sizeof(size_t), &width);
	createDeviceVar((void **)&d_height, sizeof(size_t), &height);
	createDeviceVar((void **)&d_trap, sizeof(quadrilateral_t), &trap);

	// setting up processing specifics
	dim3 threads(512, 1); // number of threads per block
	dim3 blocks(width * height / threads.x, 1); // number of blocks in data set to be calculated

	// setup start stop events
	hipEvent_t start, stop;
	checkCudaErrors(hipEventCreate(&start));
	checkCudaErrors(hipEventCreate(&stop));

	hipEventRecord(start, 0); // start event trigger
	transform<<<blocks, threads>>>(d_col_in, d_col_out, d_width, d_height, d_trap); // perform calculation
	hipMemcpyAsync(tf_cols, d_col_out, 4 * sizeof(sf::Uint8) * width * height, hipMemcpyDeviceToHost, 0); // copy useful data back

	hipEventRecord(stop, 0); // stop event trigger

	while (hipEventQuery(stop) == hipErrorNotReady) // wait for stop
	{
			usleep(100);
	}

	// cleaning up
	checkCudaErrors(hipEventDestroy(start));
	checkCudaErrors(hipEventDestroy(stop));
	checkCudaErrors(hipFree(d_col_in));
	checkCudaErrors(hipFree(d_col_out));
	checkCudaErrors(hipFree(d_width));
	checkCudaErrors(hipFree(d_height));
	checkCudaErrors(hipFree(d_trap));
}
