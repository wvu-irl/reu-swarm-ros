#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <unistd.h>
#include <iostream>
#include <tuple>

#include <hip/hip_runtime.h>
#include "inc/hip/hip_runtime_api.h"
#include "inc/helper_functions.h"

#include <SFML/Graphics.hpp>
#include <functional>
#include <math.h>
#include "color_map.h"

/**
 * namespace for doing stupid amounts of calculations
 */
namespace hyperthread
{

// device details
int devID;
hipDeviceProp_t deviceProp;

/**
 * 4d surface function to be calculated
 *
 * z is the output of the direction "towards the camera"
 * x and y are coordinates on the plane perpendicular to the view
 * t is a saw function of time (goes from 0 to 1000 incrementing by 1 every tick)
 */
__device__ void zfunc(double *z, double x, double y, double t)
{
    x -= 640;
		y -= 400;
		x /= 50;
		y /= 50;
		//*z = (x * x - y * y) * sin(3.14 / 500 * t);
		double r0 = sqrt(x*x + y*y);
		*z = 20 * pow(M_E, -pow(r0, 2) / (2 * pow(sin(M_PI / 200 * t), 2)));

		y -= 3;
		double r1 = sqrt(x*x + y*y);
		*z += 20 * pow(M_E, -pow(r1, 2) / (2 * pow(sin(M_PI / 200 * t), 2)));

		y += 6;
		double r2 = sqrt(x*x + y*y);
		*z += 20 * pow(M_E, -pow(r2, 2) / (2 * pow(sin(M_PI / 200 * t), 2)));
}

/*
 * Internal color struct for use by CUDA functions
 *
 * transferring sf::Color was not possible
 */
typedef struct
{
    sf::Uint8 r;
    sf::Uint8 g;
    sf::Uint8 b;
} color;

/**
 * Scales the value val from its original range to a new range
 *
 * val is the input value in the range of o_min to o_max
 * o_min and o_max is the start range
 * n_min and n_max is the output range
 *
 * returns the value scaled to the new range
 *
 */
__device__ double scale(double val, double o_min, double o_max, double n_min,double n_max)
{
    return (val - o_min) / (o_max - o_min) * (n_max - n_min) + n_min; // behold it is math
}

/**
 * Translated function from color map to run on __device__
 *
 * calculates the color in the specified gradient
 *
 * val is the value within the gradient
 * colors are the colors that are part of the gradient
 * color_levels are the respective 'levels' the colors are on
 * 							- Levels are z values for the color that the value is approching
 * num_cols is the nubmer of colors in both colors and color_levels
 *
 */
__device__ color calculateColor(double val, color colors[], double color_levels[], size_t num_cols)
{
	// low edge case
	if (val <= color_levels[0])
		return colors[0];

	// high edge case
	if (val >= color_levels[num_cols - 1])
		return colors[num_cols - 1];

	// mid case
	for (size_t i = 1; i < num_cols; i++)
	{
		// finding the first color to have a greater value than the input
		if (color_levels[i] > val)
		{
			color col;
			// finding scaled RGB value
			col.r = (int) scale(val, color_levels[i], color_levels[i - 1], colors[i].r,
							colors[i - 1].r);
			col.g = (int) scale(val, color_levels[i], color_levels[i - 1], colors[i].g,
							colors[i - 1].g);
			col.b = (int) scale(val, color_levels[i], color_levels[i - 1], colors[i].b,
							colors[i - 1].b);
			return col;
		}
	}
	return (color){127,127,127};
}

// finds a maximum from device
__device__ double max(double a, double b)
{
    return a > b ? a : b;
}

// finds a minimum from device
__device__ double min(double a, double b)
{
    return a > b ? b : a;
}

/**
 * Calculates a 4d surface function contour and heat plot
 *
 * levels is an array of desired contour levels
 * num_levels is the number of contour levels
 *
 * cols is an empty image set for an sf::Image
 * width and height are the dumensions of the screen
 *
 * colors is an array of colors from a ColorMap used in calculateColor
 * color_levels is the respective levels used in the ColorMap
 * num_cols is the number of color levels
 *
 * t is the current tick from the tick saw function
 *
 */
__global__ void gpuThread(double *levels, size_t *num_levels, sf::Uint8 *cols,
    size_t *width, size_t *height, color *colors, double *color_levels, size_t *num_cols, int *t)
{
		// finding where in the image the process is
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int col_id = idx * 4;
    int i = idx / (*width);
    int j = idx % (*width);

    // calculating function
    double zc;
    zfunc(&zc, (double)j, (double)i, (double)(*t));

    // determining gradiant color
    color c = calculateColor(zc, colors, color_levels, *num_cols);
    cols[col_id] = c.r;
    cols[col_id + 1] = c.g;
    cols[col_id + 2] = c.b;
    cols[col_id + 3] = 255;

    // checking if there are contours to draw
    if (num_levels > 0)
    {
    		// calculating neighbors
        double zz[4];
        zfunc(zz, (double)j, (double)i - 1, (double)(*t));
        zfunc(zz + 1, (double)j, (double)i + 1, (double)(*t));
        zfunc(zz + 2, (double)j - 1, (double)i, (double)(*t));
        zfunc(zz + 3, (double)j + 1, (double)i, (double)(*t));

        // checking levels
        for (size_t k = 0;k < *num_levels;k++)
        {
        		// checking if a level is crossed
            bool draw = false;
            for (int l = 0;l < 4 && !draw;l++)
            {
                if (min(zc, zz[l]) <= levels[k] && levels[k] <= max(zc, zz[l]))
                {
                    draw = true;
                }
            }

            // drawing level if needed
            if (draw)
            {
                cols[col_id] = 0;       // R
                cols[col_id + 1] = 0;   // G
                cols[col_id + 2] = 0;   // B
                cols[col_id + 3] = 0;   // A
                break; // getting out of loop because line was drawn
            }
        }
    }
}

// helper function to copy variables to pointers for the device
void createDeviceVar(void **var, size_t size, void *h_var)
{
    std::cout << "\033[31;1m" << std::flush;
    checkCudaErrors(hipMalloc(var, size));
    checkCudaErrors(hipMemset(*var, 0, size));
    std::cout << "\033[0m" << std::flush;

    hipMemcpyAsync(*var, h_var, size, hipMemcpyHostToDevice, 0);
}

/**
 * Calculates all the information for a contour plot through the GPU
 *
 * cols is a pointer to the image data
 * levels is a vector of all the contour levels
 * width and height are the dimensions of the window
 *
 * colors color_levels num_cols are a separation of relevant data from a ColorMap
 *
 * t is the tick saw function
 *
 */
void calc(sf::Uint8 *cols, std::vector<double> levels, size_t width,
    size_t height, color colors[], double color_levels[], size_t num_cols, int t)
{
    size_t n = width * height;
    size_t size = n * 4;

    // setting up device pointers
    sf::Uint8 *d_cols;

    double *d_levels;
    size_t *d_num_levels;

    size_t *d_width, *d_height;

    color *d_colors;
    double *d_color_levels;
    size_t *d_num_colors;

    int *d_time;

    size_t n_levels = levels.size();

    createDeviceVar((void **)&d_cols, size, cols);
    createDeviceVar((void **)&d_levels, sizeof(double) * levels.size(), levels.data());
    createDeviceVar((void **)&d_num_levels, sizeof(size_t), &n_levels);
    createDeviceVar((void **)&d_width, sizeof(width), &width);
    createDeviceVar((void **)&d_height, sizeof(height), &height);
    createDeviceVar((void **)&d_colors, sizeof(color) * num_cols, colors);
    createDeviceVar((void **)&d_color_levels, sizeof(double) * num_cols, color_levels);
    createDeviceVar((void **)&d_num_colors, sizeof(size_t), &(num_cols));
    createDeviceVar((void **)&d_time, sizeof(int), &t);

    dim3 threads(1024, 1);
    dim3 blocks(n / threads.x, 1);

    hipEvent_t start, stop;
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));

    hipEventRecord(start, 0);
    gpuThread<<<blocks, threads>>>(
        d_levels,
        d_num_levels,
        d_cols,
        d_width,
        d_height,
        d_colors,
        d_color_levels,
        d_num_colors,
        d_time);
    hipMemcpyAsync(cols, d_cols, size, hipMemcpyDeviceToHost, 0);
    hipEventRecord(stop, 0);

    while (hipEventQuery(stop) == hipErrorNotReady)
    {
        usleep(100);
    }

    std::cout << "\033[31;1m" << std::flush;
    checkCudaErrors(hipEventDestroy(start));
    checkCudaErrors(hipEventDestroy(stop));
    checkCudaErrors(hipFree(d_cols));
    checkCudaErrors(hipFree(d_levels));
    checkCudaErrors(hipFree(d_num_levels));
    checkCudaErrors(hipFree(d_width));
    checkCudaErrors(hipFree(d_height));
    checkCudaErrors(hipFree(d_colors));
    checkCudaErrors(hipFree(d_color_levels));
    checkCudaErrors(hipFree(d_num_colors));
    checkCudaErrors(hipFree(d_time));
    std::cout << "\033[0m" << std::flush;
}

void init()
{
    const char *str = "";
    devID = findCudaDevice(0, &str);
    checkCudaErrors(hipGetDeviceProperties(&deviceProp, devID));
}

void destruct()
{
}

}
