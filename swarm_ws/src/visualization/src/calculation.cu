#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <unistd.h>
#include <iostream>
#include <tuple>

#include <hip/hip_runtime.h>
#include "inc/hip/hip_runtime_api.h"
#include "inc/helper_functions.h"

#include <SFML/Graphics.hpp>
#include <functional>
#include "color_map.h"

#define DEBUG_GPU_CU 1


namespace hyperthread
{

int devID;
hipDeviceProp_t deviceProp;

sf::Uint8 *h_cols;
bool allocated = false;

__device__ void zfunc(double *z, double x, double y, double t)
{
    x -= 640;
	y -= 400;
	x /= 50;
	y /= 50;
	*z = (x * x - y * y) * sin(3.14 / 500 * t);
}

typedef struct
{
    sf::Uint8 r;
    sf::Uint8 g;
    sf::Uint8 b;
} color;

__device__ double scale(double val, double o_min, double o_max, double n_min,double n_max)
{
    return (val - o_min) / (o_max - o_min) * (n_max - n_min) + n_min;
}

__device__ color calculateColor(double val, color colors[], double color_levels[], size_t num_cols)
{
	// low edge case
	if (val <= color_levels[0])
		return colors[0];

	// high edge case
	if (val >= color_levels[num_cols - 1])
		return colors[num_cols - 1];

	// mid case
	for (size_t i = 1; i < num_cols; i++)
	{
		// finding the first color to have a greater value than the input
		if (color_levels[i] > val)
		{
			color col;
			// finding scaled RGB value
			col.r = (int) scale(val, color_levels[i], color_levels[i - 1], colors[i].r,
							colors[i - 1].r);
			col.g = (int) scale(val, color_levels[i], color_levels[i - 1], colors[i].g,
							colors[i - 1].g);
			col.b = (int) scale(val, color_levels[i], color_levels[i - 1], colors[i].b,
							colors[i - 1].b);
			return col;
		}
	}
	return (color){127,127,127};
}

__device__ double max(double a, double b)
{
    return a > b ? a : b;
}

__device__ double min(double a, double b)
{
    return a > b ? b : a;
}

__global__ void gpuThread(double *levels, size_t *num_levels, sf::Uint8 *cols, 
    size_t *width, size_t *height, color *colors, double *color_levels, size_t *num_cols, int *t)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int col_id = idx * 4;
    int i = idx / (*width);
    int j = idx % (*width);

    double zc;
    zfunc(&zc, (double)j, (double)i, (double)(*t));
    color c = calculateColor(zc, colors, color_levels, *num_cols);
    cols[col_id] = c.r;
    cols[col_id + 1] = c.g;
    cols[col_id + 2] = c.b;
    cols[col_id + 3] = 255;

    if (num_levels > 0)
    {
        double zz[4];
        zfunc(zz, (double)j, (double)i - 1, (double)(*t));
        zfunc(zz + 1, (double)j, (double)i + 1, (double)(*t));
        zfunc(zz + 2, (double)j - 1, (double)i, (double)(*t));
        zfunc(zz + 3, (double)j + 1, (double)i, (double)(*t));

        for (size_t k = 0;k < *num_levels;k++)
        {
            bool draw = false;
            for (int l = 0;l < 4 && !draw;l++)
            {
                if (min(zc, zz[l]) <= levels[k] && levels[k] <= max(zc, zz[l]))
                {
                    draw = true;   
                }
            }

            if (draw)
            {
                cols[col_id] = 0;       // R
                cols[col_id + 1] = 0;   // G
                cols[col_id + 2] = 0;   // B
                cols[col_id + 3] = 0;   // A
                break;
            }
        }
    }
}

void createDeviceVar(void **var, size_t size, void *h_var)
{
    std::cout << "\033[31;1m" << std::flush;
    checkCudaErrors(hipMalloc(var, size));
    checkCudaErrors(hipMemset(*var, 0, size));
    std::cout << "\033[0m" << std::flush;

    hipMemcpyAsync(*var, h_var, size, hipMemcpyHostToDevice, 0);
}

void calc(sf::Uint8 *cols, std::vector<double> levels, size_t width, 
    size_t height, color colors[], double color_levels[], size_t num_cols, int t)
{
    size_t n = width * height;
    size_t size = n * 4;

    // setting up device pointers
    sf::Uint8 *d_cols;
    
    double *d_levels;
    size_t *d_num_levels;
    
    size_t *d_width, *d_height;
    
    color *d_colors;
    double *d_color_levels;
    size_t *d_num_colors;

    int *d_time;

    size_t n_levels = levels.size();

    createDeviceVar((void **)&d_cols, size, cols);
    createDeviceVar((void **)&d_levels, sizeof(double) * levels.size(), levels.data());
    createDeviceVar((void **)&d_num_levels, sizeof(size_t), &n_levels);
    createDeviceVar((void **)&d_width, sizeof(width), &width);
    createDeviceVar((void **)&d_height, sizeof(height), &height);
    createDeviceVar((void **)&d_colors, sizeof(color) * num_cols, colors);
    createDeviceVar((void **)&d_color_levels, sizeof(double) * num_cols, color_levels);
    createDeviceVar((void **)&d_num_colors, sizeof(size_t), &(num_cols));
    createDeviceVar((void **)&d_time, sizeof(int), &t);

    dim3 threads(1024, 1);
    dim3 blocks(n / threads.x, 1);

    hipEvent_t start, stop;
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));

    hipEventRecord(start, 0);
    gpuThread<<<blocks, threads>>>(
        d_levels,
        d_num_levels,
        d_cols, 
        d_width, 
        d_height, 
        d_colors, 
        d_color_levels, 
        d_num_colors,
        d_time);
    hipMemcpyAsync(cols, d_cols, size, hipMemcpyDeviceToHost, 0);
    hipEventRecord(stop, 0);

    while (hipEventQuery(stop) == hipErrorNotReady)
    {
        usleep(100);
    }

    std::cout << "\033[31;1m" << std::flush;
    checkCudaErrors(hipEventDestroy(start));
    checkCudaErrors(hipEventDestroy(stop));
    checkCudaErrors(hipFree(d_cols));
    checkCudaErrors(hipFree(d_levels));
    checkCudaErrors(hipFree(d_num_levels));
    checkCudaErrors(hipFree(d_width));
    checkCudaErrors(hipFree(d_height));
    checkCudaErrors(hipFree(d_colors));
    checkCudaErrors(hipFree(d_color_levels));
    checkCudaErrors(hipFree(d_num_colors));
    checkCudaErrors(hipFree(d_time));
    std::cout << "\033[0m" << std::flush;
}

void init()
{
    const char *str = "";
    devID = findCudaDevice(0, &str);
    checkCudaErrors(hipGetDeviceProperties(&deviceProp, devID));
}

void destruct()
{
}

}